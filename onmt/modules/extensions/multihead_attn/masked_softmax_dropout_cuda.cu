#include <vector>
#include <iostream>

#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include "THC/THC.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <math.h>

#include "softmax.h"
#include "dropout.h"
#include "additive_time_masked_softmax.h"

// symbol to be automatically resolved by PyTorch libs
extern THCState *state;

namespace multihead_attn {
namespace fused_softmax {
namespace mask_softmax_dropout {

std::vector<torch::Tensor> fwd_cuda(
			                   bool                 is_training,
			                   bool                 time_mask,
                               int                  heads,
                               torch::Tensor const& input, 
                               const half*        pad_mask,
                               float                dropout_prob
                                   )
{
  const int   attn_batches   = input.size(0);
  const int   sequences      = attn_batches / heads;
  const int   q_seq_len      = input.size(1);
  const int   k_seq_len      = input.size(2);
  const int   dropout_elems  = attn_batches * q_seq_len * k_seq_len;

  // There is no reason to use more than one stream as every kernel is 
  // sequentially dependent
  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  hipStream_t   stream = at::cuda::getCurrentCUDAStream().stream();
  hipblasSetStream(handle, stream);

  auto act_options  = input.options().requires_grad(false);
  auto mask_options = act_options.dtype(torch::kUInt8);

  torch::Tensor softmax_results   = torch::empty({attn_batches, q_seq_len, k_seq_len},   act_options);
//  torch::Tensor dropout_results   = torch::empty({attn_batches, q_seq_len, k_seq_len},   act_options);
  torch::Tensor dropout_mask      = torch::empty({attn_batches, q_seq_len, k_seq_len},   mask_options);

  // Softmax Intermediate Result Ptr (used by Matmul1 -> Softmax)
  void* input_ptr = static_cast<void*>(input.data_ptr());

  float dropout_keep_prob = 1.0 - dropout_prob;
  if (is_training) {
    dropout_keep_prob = 1.0;
  }

  // Padded Softmax
  bool softmax_success = false;

  void* softmax_results_ptr = static_cast<void*>(softmax_results.data_ptr());

  if (time_mask) {

  softmax_success = dispatch_additive_time_masked_softmax_dropout<half, half, float>(
                             reinterpret_cast<half*>(softmax_results_ptr),
                             reinterpret_cast<uint8_t*>(dropout_mask.data_ptr()),
                             reinterpret_cast<const half*>(input_ptr),
                             pad_mask,
                             dropout_elems,
                             k_seq_len,
                             k_seq_len,
                             attn_batches*q_seq_len,
                             q_seq_len,
                             dropout_keep_prob,
                             stream);

  }
  else {

    softmax_success = dispatch_additive_masked_softmax_dropout<half, half, float>(
                             reinterpret_cast<half*>(softmax_results_ptr),
                             reinterpret_cast<uint8_t*>(dropout_mask.data_ptr()),
                             reinterpret_cast<const half*>(input_ptr),
                             pad_mask,
                             dropout_elems,
                             k_seq_len,
                             k_seq_len,
                             attn_batches*q_seq_len,
                             attn_batches*q_seq_len/sequences,
                             dropout_keep_prob,
                             stream);
  }

  assert(softmax_success);

  return {
           dropout_mask, 
           softmax_results
         };
}

torch::Tensor bwd_cuda(
		               int heads,
                       torch::Tensor const& output_grads,
                       torch::Tensor const& softmax_results,
                       torch::Tensor const& dropout_mask,
                       float                dropout_prob
                       )
{
  const int   attn_batches   = output_grads.size(0);
  const int   q_seq_len      = output_grads.size(1);
  const int   k_seq_len      = output_grads.size(2);
  const int   dropout_elems  = attn_batches * q_seq_len * k_seq_len;
  // TODO: Streams can be used in Backprop but I haven't added more than one
  // in my first attempt to create the code
  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  hipStream_t   stream = at::cuda::getCurrentCUDAStream().stream();
  hipblasSetStream(handle, stream);

  // Output Tensor Allocations
//  torch::Tensor input_grads         = torch::empty_like(output_grads);

  // Apply Dropout Mask and Scale by Dropout Probability 
  // Softmax Grad
//  if (padding_mask == nullptr) {
//      dispatch_masked_scale_softmax_backward_stream<half, half, float,false>(
//                             static_cast<half*>(output_grads.data_ptr()),
//                             static_cast<half*>(output_grads.data_ptr()),
//                             reinterpret_cast<half const*>(softmax_results.data_ptr()),
//			     static_cast<uint8_t const*>(dropout_mask.data_ptr()),
//			     1.0/(1.0-dropout_prob),
//                             k_seq_len,
//                             k_seq_len,
//                             attn_batches*q_seq_len, stream);
//  } else{
//      dispatch_masked_scale_softmax_backward_masked_out_stream<half, half, float,false>(
//                             static_cast<half*>(output_grads.data_ptr()),
//                             static_cast<half*>(output_grads.data_ptr()),
//                             reinterpret_cast<half const*>(softmax_results.data_ptr()),
//			     static_cast<uint8_t const*>(dropout_mask.data_ptr()),
//			     static_cast<uint8_t const*>(padding_mask),
//			     1.0/(1.0-dropout_prob),
//                             k_seq_len,
//                             k_seq_len,
//                             attn_batches*q_seq_len,
//			     heads, stream);
  dispatch_masked_scale_softmax_backward_stream<half, half, float,false>(
                             static_cast<half*>(output_grads.data_ptr()),
                             static_cast<half*>(output_grads.data_ptr()),
                             reinterpret_cast<half const*>(softmax_results.data_ptr()),
                             static_cast<uint8_t const*>(dropout_mask.data_ptr()),
                             1.0/(1.0-dropout_prob),
                             k_seq_len,
                             k_seq_len,
                             attn_batches*q_seq_len, stream);

//  // alternatively
//  // Apply Dropout Mask and Scale by Dropout Probability
//  apex_masked_scale_cuda<at::Half,float,uint32_t>(
//                             static_cast<at::Half const*>(output_grads.data_ptr()),
//                             static_cast<at::Half*>(output_grads.data_ptr()),
//                             static_cast<uint8_t const*>(dropout_mask.data_ptr()),
//                             dropout_elems,
//                             (1.0 / (1.0 - dropout_prob)));
////
////  // Softmax Grad
////  bool softmax_success = false;
//  softmax_success = dispatch_softmax_backward<half, half, float>(
//                             static_cast<half*>(output_grads.data_ptr()),
//                             static_cast<half*>(output_grads.data_ptr()),
//                             reinterpret_cast<half const*>(softmax_results.data_ptr()),
//                             k_seq_len,
//                             k_seq_len,
//                             attn_batches*q_seq_len);


  //backward pass is completely in-place
  return output_grads;
}
}
}
}

